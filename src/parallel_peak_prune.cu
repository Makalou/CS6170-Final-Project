#include "hip/hip_runtime.h"
#include "parallel_peak_prune.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_partition.cuh>

__global__ 
void assign_init_label_kernel(PPPEdge* edges, PPPVertex* vertices, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        PPPEdge e = edges[index];
        int lower_end;
        int higher_end;
        if (vertices[e.v1].val > vertices[e.v2].val)
        {
            higher_end = e.v1;
            lower_end = e.v2;
        }
        else {
            higher_end = e.v2;
            lower_end = e.v1;
        }//todo what if vertices[e.v1].val == vertices[e.v2].val?

        //https://stackoverflow.com/questions/52848426/how-to-execute-atomic-write-in-cuda
            //We don't need atomic operation here
        vertices[lower_end].peak_label = higher_end;

        edges[index].v1 = lower_end;
        edges[index].v2 = higher_end;
    }
}

void assign_init_label(PPPEdge* d_edges, PPPVertex* d_vertices, int size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    assign_init_label_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_edges, d_vertices,size);
    hipDeviceSynchronize();
}

__global__
void pointer_jump_kernel1(const PPPVertex* vertices, int* peak_labels, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        peak_labels[index] = vertices[index].peak_label;
    }
}

__global__
void pointer_jump_kernel2(PPPVertex* vertices, const int* peak_labels, int size, int* d_counter)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        int peak = peak_labels[index];
        //todo device level sychronize need
        if (peak != vertices[peak].peak_label)
        {
            vertices[index].peak_label = vertices[peak].peak_label;
            atomicAdd(d_counter, 1);
        }
    }
}

void pointer_jump(PPPVertex* d_vertices, int* d_aux_labels, int size, int* d_counter)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    // Read the preceder's label to labels buffer
    pointer_jump_kernel1 <<<blocksPerGrid, threadsPerBlock >>> (d_vertices,d_aux_labels, size);
    // Write preceder's label to vertex label, perform one pass pointer jump.
    // The separation of these two procedure is neccessary to ensure correct synchronization.
    hipDeviceSynchronize();
    pointer_jump_kernel2<<<blocksPerGrid, threadsPerBlock>>>(d_vertices,d_aux_labels, size, d_counter);
    hipDeviceSynchronize();
}

__global__
void compact_peak_label_kernel1(const PPPVertex* vertices, PPPLabelIndex* labelIdx, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        labelIdx[index].label = vertices[index].peak_label;
        labelIdx[index].idx = index;
    }
}

__global__
void compact_peak_label_kernel2(const PPPLabelIndex* labelIdx,int* flags,int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (index != 0)
        {
            if (labelIdx[index - 1].label != labelIdx[index].label)
            {
                flags[index] = 0;
            }
            else {
                flags[index] = 1;
            }
        }
    }
}

__global__
void compact_peak_label_kernel3(PPPVertex* vertices, const PPPLabelIndex* labelIdx,const int* flags, int* compact_labels_map,int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (index == 0 || flags[index - 1] != flags[index])
        {
            // Store orginal peak_label to compacted label idx
           compact_labels_map[flags[index]] = vertices[labelIdx[index].idx].peak_label;
        }
        vertices[labelIdx[index].idx].peak_label = flags[index];
    }
}

void count_peaks(PPPVertex* vertices, PPPLabelIndex* d_label_idx, int* d_flags1, int* d_flags2, int size, int * peak_count)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    compact_peak_label_kernel1 << <blocksPerGrid, threadsPerBlock >> > (vertices, d_label_idx, size);
    hipDeviceSynchronize();
    // sort label idx
    PPPLabelIndexComp compOp;
    hipcub::DeviceMergeSort::SortKeys(nullptr, temp_storage_bytes, d_label_idx, size, compOp);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceMergeSort::SortKeys(d_temp_storage, temp_storage_bytes, d_label_idx, size, compOp);

    compact_peak_label_kernel2 << <blocksPerGrid, threadsPerBlock >> > (d_label_idx, d_flags1, size);
    hipDeviceSynchronize();
    // inclusive scan on flags
    size_t temp_storage_bytes2 = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes2, d_flags1, d_flags2, size);
    if (temp_storage_bytes2 > temp_storage_bytes)
    {
        //Need to reallocate temp storage
        hipFree(d_temp_storage);
        hipMalloc(&d_temp_storage, temp_storage_bytes2);
    }
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes2, d_flags1, d_flags2, size);

    hipMemcpy(peak_count, &d_flags2[size - 1], sizeof(int), hipMemcpyDeviceToHost);
}

void assign_compact_peak_label(PPPVertex* vertices, PPPLabelIndex* d_label_idx,int* d_flags, int* d_compact_labels_map, int size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    compact_peak_label_kernel3<<<blocksPerGrid, threadsPerBlock>>>(vertices, d_label_idx, d_flags, d_compact_labels_map, size);
    hipDeviceSynchronize();
}

__global__
void identify_saddle_candidate_kernel(PPPEdge* edges, const PPPVertex* vertices, int* neighbor_labels, int* is_candidate, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        PPPEdge e = edges[index];
        int higher_end = e.v1;
        int lower_end = e.v2;
        // Remeber the peak label here is compacted label
        int cur = vertices[higher_end].peak_label;
        edges[index].label = cur;
        int old = atomicExch(&neighbor_labels[lower_end], cur);
        if (old != -1 && old != cur)
        {
            is_candidate[lower_end] = 1;
        }
    }
}

void identify_saddle_candidate(PPPEdge* d_edges, const PPPVertex* d_vertices, int* d_neighbor_labels, int* is_candidate, int size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    identify_saddle_candidate_kernel <<<blocksPerGrid, threadsPerBlock >>> (d_edges, d_vertices, d_neighbor_labels, is_candidate, size);
    hipDeviceSynchronize();
}

void partition_saddle_candidate_edges(const PPPEdge* edges,PPPEdge* out_edges, int* is_candidate, int size, int * selected_out)
{
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DevicePartition::Flagged(nullptr, temp_storage_bytes, edges, is_candidate, out_edges, selected_out, size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, edges, is_candidate, out_edges, selected_out, size);
    hipFree(d_temp_storage);
}

void sort_saddle_candidate_edges(PPPEdge* d_edges, const PPPVertex* d_vertices, int size)
{
    // sort candidate edges by (peak label, lower end)
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    PPPEdgeLess less_op(d_vertices);
    hipcub::DeviceMergeSort::SortKeys(nullptr, temp_storage_bytes, d_edges, size, less_op);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceMergeSort::SortKeys(d_temp_storage, temp_storage_bytes, d_edges, size, less_op);
    hipFree(d_temp_storage);
}

__global__
void identify_saddle_kernel(const PPPEdge* d_edges, const PPPVertex* d_vertices, int* peak_saddle_pairs, int size, int* d_saddle_count)
{
    // Todo : can we make sure that each peak only has one governing saddle so that no write conflit would happen?
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        // Remember here the peak label is compacted label
        if (index == 0)
        {
            // The lower end of d_edges[0] is governing saddle point
            peak_saddle_pairs[d_vertices[d_edges[0].v1].peak_label] = d_edges[0].v1;
            atomicAdd(d_saddle_count, 1);
        }
        else {
            if (d_vertices[d_edges[index - 1].v1].peak_label != d_vertices[d_edges[index].v1].peak_label)
            {
                // The lower end of d_edges[index] is governing saddle point
                peak_saddle_pairs[d_vertices[d_edges[index].v1].peak_label] = d_edges[index].v1;
                atomicAdd(d_saddle_count, 1);
            }
        }
    }
}

void identify_governing_saddle(const PPPEdge* d_candidate_edges, const PPPVertex* d_vertices, int* peak_saddle_pairs, int size, int* saddle_count)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int* d_saddle_count;
    hipMalloc((void**)&d_saddle_count, sizeof(int));
    hipMemset(d_saddle_count, 0, sizeof(int));
    identify_saddle_kernel <<<blocksPerGrid, threadsPerBlock>>> (d_candidate_edges, d_vertices, peak_saddle_pairs, size, d_saddle_count);
    hipDeviceSynchronize();
    hipMemcpy(saddle_count, d_saddle_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_saddle_count);
}

__global__
void mark_delete_vertices_kernel(const PPPVertex* d_vertices, const int* peak_saddle_pairs, int* vet_should_remain, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        const auto v = d_vertices[index];
        if (v.val <= d_vertices[peak_saddle_pairs[v.peak_label]].val)
        {
            vet_should_remain[index] = 1;
        }
    }
}

__global__
void get_vet_reorder_map_kernel(const int * d_vet_remain_scan, const int* vet_should_remain, int* d_vet_reorder_map, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        if (vet_should_remain[index] > 0)
        {
            d_vet_reorder_map[index] = d_vet_remain_scan[index];
        }
    }
}

__global__
void mark_delete_edges_kernel(const PPPEdge* d_edges,const int* vet_should_remain, int* edge_should_remain, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        const auto e = d_edges[index];
        if (vet_should_remain[e.v1] > 0 && vet_should_remain[e.v2] > 0)
        {
            edge_should_remain[index] = 1;
        }
    }
}

__global__
void edges_redirect_kernel(PPPEdge* d_edges, const int* vet_should_remain, 
                           const int* vet_new_idx_map, const int* peak_saddle_pairs, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        auto e = d_edges[index];
        e.v1 = vet_new_idx_map[e.v1];
        if (vet_should_remain[e.v2] > 0)
        {
            // If the higher end of the edge is deleted, redirect it to the governing saddle
            e.v2 = vet_new_idx_map[peak_saddle_pairs[e.label]];
        }
        else {
            e.v2 = vet_new_idx_map[e.v2];
        }
        d_edges[index] = e;
    }
}

void mark_delete_region(PPPVertex* d_vertices, PPPEdge* d_edges,
                        const int* d_peak_saddle_pairs,
                        int* d_vet_should_remain, int* d_edge_should_remain,
                        int v_size, int e_size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (v_size + threadsPerBlock - 1) / threadsPerBlock;

    mark_delete_vertices_kernel << <blocksPerGrid, threadsPerBlock >> > (d_vertices, d_peak_saddle_pairs, d_vet_should_remain, v_size);
    hipDeviceSynchronize();

    blocksPerGrid = (e_size + threadsPerBlock - 1) / threadsPerBlock;
    mark_delete_edges_kernel << <blocksPerGrid, threadsPerBlock >> > (d_edges, d_vet_should_remain, d_edge_should_remain, e_size);
    hipDeviceSynchronize();
}

void flatten_vertices_and_edges(const PPPVertex* d_vertices, const PPPEdge* d_edges,
                                PPPVertex* d_new_vertices, PPPEdge* d_new_edges,
                                const int* d_vet_should_remain, const int* d_edge_should_remain,
                                int* d_vet_remain_scan, int* d_vet_reorder_map,
                                int v_size, int e_size, int* remain_v_size, int* remain_e_size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (v_size + threadsPerBlock - 1) / threadsPerBlock;

    int* d_remain_v_size;
    int* d_remain_e_size;
    hipMalloc((void**)&d_remain_v_size, sizeof(int));
    hipMalloc((void**)&d_remain_e_size, sizeof(int));
    // Reorder the vertices, get vertex new index map, get the size of new vertice set
    // We have to assume hipcub::DevicePartition is stable
    // d_vet_reorder_map stores the target position for *remained* vertices after partition
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DevicePartition::Flagged(nullptr, temp_storage_bytes, d_vertices, d_vet_should_remain, d_new_vertices, d_remain_v_size, v_size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes, d_vertices, d_vet_should_remain, d_new_vertices, d_remain_v_size, v_size);
    hipMemcpy(remain_v_size, d_remain_v_size, sizeof(int), hipMemcpyDeviceToHost);

    blocksPerGrid = (v_size + threadsPerBlock - 1) / threadsPerBlock;
    size_t temp_storage_bytes2 = 0;
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes2, d_vet_should_remain, d_vet_remain_scan, v_size);
    if (temp_storage_bytes2 > temp_storage_bytes2)
    {
        hipFree(d_temp_storage);
        hipMalloc(&d_temp_storage, temp_storage_bytes2);
        temp_storage_bytes = temp_storage_bytes2;
    }
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes2, d_vet_should_remain, d_vet_remain_scan, v_size);

    get_vet_reorder_map_kernel << <blocksPerGrid, threadsPerBlock >> > (d_vet_remain_scan, d_vet_should_remain, d_vet_reorder_map, *remain_v_size);
    hipDeviceSynchronize();

    // Reorder the edges, get the size of new edge set
    size_t temp_storage_bytes3 = 0;
    hipcub::DevicePartition::Flagged(nullptr, temp_storage_bytes3, d_edges, d_edge_should_remain, d_new_edges, d_remain_e_size, e_size);
    if (temp_storage_bytes3 > temp_storage_bytes)
    {
        hipFree(d_temp_storage);
        hipMalloc(&d_temp_storage, temp_storage_bytes3);
        temp_storage_bytes = temp_storage_bytes3;
    }
    hipcub::DevicePartition::Flagged(d_temp_storage, temp_storage_bytes3, d_edges, d_edge_should_remain, d_new_edges, d_remain_e_size, e_size);
    hipMemcpy(remain_e_size, d_remain_e_size, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_temp_storage);
    hipFree(d_remain_v_size);
    hipFree(d_remain_e_size);
}

void redirect_edges(PPPEdge* d_edges, int* d_vet_should_remain, int* d_vet_new_idx_map, const int* d_peak_saddle_pairs, int size)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    edges_redirect_kernel << <blocksPerGrid, threadsPerBlock >> > (d_edges, d_vet_should_remain, d_vet_new_idx_map, d_peak_saddle_pairs, size);
    hipDeviceSynchronize();
}